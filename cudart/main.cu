#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
#define NUM_SPHERES 488
#define NUM_LAM_SPHERES 2
#define NUM_MET_SPHERES 1
#define NUM_DIE_SPHERES 485

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ bool sphere_hit(const sphere &s, const ray& r, float t_min, float t_max, hit_record& rec) {
    vec3 oc = r.origin() - s.center;
    float a = dot(r.direction(), r.direction());
    float b = dot(oc, r.direction());
    float c = dot(oc, oc) - s.radius*s.radius;
    float discriminant = b*b - a*c;
    if (discriminant > 0) {
        float temp = (-b - sqrt(discriminant))/a;
        if (temp < t_max && temp > t_min) {
            rec.t = temp;
            rec.p = r.point_at_parameter(rec.t);
            rec.normal = (rec.p - s.center) / s.radius;
            rec.mat_ptr = s.mat_ptr;
            return true;
        }
        temp = (-b + sqrt(discriminant)) / a;
        if (temp < t_max && temp > t_min) {
            rec.t = temp;
            rec.p = r.point_at_parameter(rec.t);
            rec.normal = (rec.p - s.center) / s.radius;
            rec.mat_ptr = s.mat_ptr;
            return true;
        }
    }
    return false;
}

__device__ bool world_hit(const ray& r, const sphere** list, int list_size, float t_min, float t_max, hit_record& rec) {
        hit_record temp_rec;
        bool hit_anything = false;
        float closest_so_far = t_max;
        for (int i = 0; i < list_size; i++) {
            if (sphere_hit((*list)[i], r, t_min, closest_so_far, temp_rec)) {
                hit_anything = true;
                closest_so_far = temp_rec.t;
                rec = temp_rec;
                if (i < NUM_LAM_SPHERES) {
                    rec.type = 0;
                }
                else if (i >= NUM_LAM_SPHERES && i < NUM_LAM_SPHERES + NUM_MET_SPHERES) {
                    rec.type = 1;
                }
                else if (i >= NUM_LAM_SPHERES + NUM_MET_SPHERES && i < NUM_LAM_SPHERES + NUM_MET_SPHERES + NUM_DIE_SPHERES) {
                    rec.type = 2;
                }
            }
        }
        return hit_anything;
}

__device__ bool lambertian_scatter(const lambertian* l, const ray& r_in, const hit_record& rec, vec3& attenuation, ray& scattered, hiprandState *local_rand_state) {
    vec3 target = rec.p + rec.normal + random_in_unit_sphere(local_rand_state);
    scattered = ray(rec.p, target-rec.p);
    attenuation = l->albedo;
    return true;
}

__device__ bool metal_scatter(const metal* m, const ray& r_in, const hit_record& rec, vec3& attenuation, ray& scattered, hiprandState *local_rand_state) {
    vec3 reflected = reflect(unit_vector(r_in.direction()), rec.normal);
    scattered = ray(rec.p, reflected + m->fuzz*random_in_unit_sphere(local_rand_state));
    attenuation = m->albedo;
    return (dot(scattered.direction(), rec.normal) > 0.0f);
}

__device__ bool dielectric_scatter(
                        const dielectric* d,
                        const ray& r_in,
                        const hit_record& rec,
                        vec3& attenuation,
                        ray& scattered,
                        hiprandState *local_rand_state) {
    vec3 outward_normal;
    vec3 reflected = reflect(r_in.direction(), rec.normal);
    float ni_over_nt;
    attenuation = vec3(1.0, 1.0, 1.0);
    vec3 refracted;
    float reflect_prob;
    float cosine;
    if (dot(r_in.direction(), rec.normal) > 0.0f) {
        outward_normal = -rec.normal;
        ni_over_nt = d->ref_idx;
        cosine = dot(r_in.direction(), rec.normal) / r_in.direction().length();
        cosine = sqrt(1.0f - d->ref_idx*d->ref_idx*(1-cosine*cosine));
    }
    else {
        outward_normal = rec.normal;
        ni_over_nt = 1.0f / d->ref_idx;
        cosine = -dot(r_in.direction(), rec.normal) / r_in.direction().length();
    }
    if (refract(r_in.direction(), outward_normal, ni_over_nt, refracted))
        reflect_prob = schlick(cosine, d->ref_idx);
    else
        reflect_prob = 1.0f;
    if (hiprand_uniform(local_rand_state) < reflect_prob)
        scattered = ray(rec.p, reflected);
    else
        scattered = ray(rec.p, refracted);
    return true;
}

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 color(const ray& r, const sphere** list, int list_size, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if (world_hit(cur_ray, list, list_size, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if (rec.type == 0 && lambertian_scatter((lambertian*)rec.mat_ptr, cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else if (rec.type == 1 && metal_scatter((metal*)rec.mat_ptr, cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else if (rec.type == 2 && dielectric_scatter((dielectric*)rec.mat_ptr, cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0,0.0,0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    // Original: Each thread gets same seed, a different sequence number, no offset
    // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
    // performance improvement of about 2x!
    hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;

    hitable_list **hitable_list_word = (hitable_list**)(world);

    extern __shared__ char array[];
    int index = 0;
    sphere *sphere_list = (sphere *)array;
    index += NUM_SPHERES*sizeof(sphere);
    lambertian* lam_list = (lambertian *)(array + index);
    index += NUM_LAM_SPHERES*sizeof(lambertian);
    metal* met_list = (metal *)(array + index);
    index += NUM_MET_SPHERES*sizeof(metal);
    dielectric* die_list = (dielectric *)(array + index);
    index += NUM_DIE_SPHERES*sizeof(dielectric);

    for (int i = 0; i < NUM_LAM_SPHERES; ++i) {
        sphere* s = (sphere*)(*((*hitable_list_word)->list + i));
        lam_list[i].albedo.e[0] = ((lambertian*)(s->mat_ptr))->albedo.e[0];
        lam_list[i].albedo.e[1] = ((lambertian*)(s->mat_ptr))->albedo.e[1];
        lam_list[i].albedo.e[2] = ((lambertian*)(s->mat_ptr))->albedo.e[2];
    }

    for (int i = NUM_LAM_SPHERES; i < NUM_LAM_SPHERES+NUM_MET_SPHERES; ++i) {
        int met_i = i - NUM_LAM_SPHERES;
        sphere* s = (sphere*)(*((*hitable_list_word)->list + i));
        met_list[met_i].albedo.e[0] = ((metal*)(s->mat_ptr))->albedo.e[0];
        met_list[met_i].albedo.e[1] = ((metal*)(s->mat_ptr))->albedo.e[1];
        met_list[met_i].albedo.e[2] = ((metal*)(s->mat_ptr))->albedo.e[2];
        met_list[met_i].fuzz = ((metal*)(s->mat_ptr))->fuzz;
    }

    for (int i = NUM_LAM_SPHERES+NUM_MET_SPHERES; i < NUM_LAM_SPHERES+NUM_MET_SPHERES+NUM_DIE_SPHERES; ++i) {
        int die_i = i - NUM_LAM_SPHERES-NUM_MET_SPHERES;
        sphere* s = (sphere*)(*((*hitable_list_word)->list + i));
        die_list[die_i].ref_idx = ((dielectric*)(s->mat_ptr))->ref_idx;
    }

    for (int i = 0; i < (*hitable_list_word)->list_size; ++i) {
        sphere* s = (sphere*)(*((*hitable_list_word)->list + i));
        sphere_list[i].center.e[0] = s->center.e[0];
        sphere_list[i].center.e[1] = s->center.e[1];
        sphere_list[i].center.e[2] = s->center.e[2];
        sphere_list[i].radius = s->radius;
        if (i < NUM_LAM_SPHERES) {
            sphere_list[i].mat_ptr = &lam_list[i];
        }
        else if (i >= NUM_LAM_SPHERES && i < NUM_LAM_SPHERES + NUM_MET_SPHERES) {
            sphere_list[i].mat_ptr = &met_list[i-NUM_LAM_SPHERES];
        }
        else if (i >= NUM_LAM_SPHERES + NUM_MET_SPHERES && i < NUM_LAM_SPHERES + NUM_MET_SPHERES + NUM_DIE_SPHERES) {
            sphere_list[i].mat_ptr = &die_list[i-NUM_LAM_SPHERES-NUM_MET_SPHERES];
        }
    }

    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, &sphere_list, (*hitable_list_word)->list_size, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0,-1000.0,-1), 1000,
                               new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for (; i < NUM_LAM_SPHERES-1; ++i) {
            int a = i / 22 - 11;
            int b = i % 22 - 11;
            vec3 center(a+RND,0.2,b+RND);
            d_list[i] = new sphere(center, 0.2,
                                     new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
        }
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        for (; i < NUM_LAM_SPHERES+NUM_MET_SPHERES-1; ++i) {
            int a = i / 22 - 11;
            int b = i % 22 - 11;
            vec3 center(a+RND,0.2,b+RND);
            d_list[i] = new sphere(center, 0.2,
                                        new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
        }
        d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        for (; i < NUM_LAM_SPHERES+NUM_MET_SPHERES+NUM_DIE_SPHERES-1; ++i) {
            int a = i / 22 - 11;
            int b = i % 22 - 11;
            vec3 center(a+RND,0.2,b+RND);
            d_list[i] = new sphere(center, 0.2, new dielectric(1.5));
        }
        d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
        *rand_state = local_rand_state;
        *d_world  = new hitable_list(d_list, NUM_SPHERES);

        vec3 lookfrom(13,2,3);
        vec3 lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    for(int i=0; i < NUM_SPHERES; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

int main() {
    int nx = 1200;
    int ny = 800;
    int ns = 10;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    hitable **d_list;
    int num_hitables = NUM_SPHERES;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads, NUM_SPHERES*sizeof(sphere)+NUM_LAM_SPHERES*sizeof(lambertian)+NUM_MET_SPHERES*sizeof(metal)+NUM_DIE_SPHERES*sizeof(dielectric)>>>(fb, nx, ny,  ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*fb[pixel_index].r());
            int ig = int(255.99*fb[pixel_index].g());
            int ib = int(255.99*fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
}
